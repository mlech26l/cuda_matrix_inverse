// includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


#include "random_matrix.h"
#include "unity_matrix.h"
#include "matrix_multiplication.h"

int main(int argc, char **argv)
{
	
	float *h_mat, *d_mat;
	int n = 12;
	
	/* Allocate n floats on host */
	h_mat = (float *)malloc(n*n* sizeof(float));
	/* Allocate n floats on device */

	d_mat = generate_random_matrix(n,100,1);
	
	/* Copy device memory to host */
	if(hipMemcpy(h_mat, d_mat, n*n * sizeof(float), hipMemcpyDeviceToHost)!= hipSuccess)
	{
		printf("Error at hipMalloc! ");
		exit(EXIT_FAILURE);
	}

	
	for(int x = 0; x < n; x++) {
		for(int y = 0; y < n; y++) {
			printf("%1.4f ", h_mat[x*n + y]);
		}
		printf("\n");
	} 
	

	float *d_b, *d_c;
	if(hipMalloc((void **)&d_b, n*n* sizeof(float)) != hipSuccess)
	{
		printf("Error on Cuda Malloc!\n");
		return NULL;
	}
	if(hipMemcpy(d_b, d_mat, n*n * sizeof(float), hipMemcpyDeviceToDevice)!= hipSuccess)
	{
		printf("Error at hipMalloc! ");
		exit(EXIT_FAILURE);
	}
	
	if(hipMalloc((void **)&d_c, n*n*sizeof(float)) != hipSuccess)
	{
		printf("Error on Cuda Malloc!\n");
		return NULL;
	}
	
	matrix_multiply(d_c,d_mat,d_mat,n);
	
	
	if(hipMemcpy(h_mat, d_c, n*n * sizeof(float), hipMemcpyDeviceToHost)!= hipSuccess)
	{
		printf("Error at hipMalloc! ");
		exit(EXIT_FAILURE);
	}
	printf("Squared matrix:\n");
	for(int x = 0; x < n; x++) {
		for(int y = 0; y < n; y++) {
			printf("%1.4f ", h_mat[x*n + y]);
		}
		printf("\n");
	} 
	
	
	float* d_unity=get_dev_unity_matrix(n);
	matrix_multiply(d_c,d_mat,d_unity,n);
	
	if(hipMemcpy(h_mat, d_unity, n*n * sizeof(float), hipMemcpyDeviceToHost)!= hipSuccess)
	{
		printf("Error at hipMalloc! ");
		exit(EXIT_FAILURE);
	}
	printf("unity matrix:\n");
	for(int x = 0; x < n; x++) {
		for(int y = 0; y < n; y++) {
			printf("%1.4f ", h_mat[x*n + y]);
		}
		printf("\n");
	} 
	
	if(hipMemcpy(h_mat, d_b, n*n * sizeof(float), hipMemcpyDeviceToHost)!= hipSuccess)
	{
		printf("Error at hipMemcpy! ");
		exit(EXIT_FAILURE);
	}
	printf("Random times unity matrix:\n");
	for(int x = 0; x < n; x++) {
		for(int y = 0; y < n; y++) {
			printf("%1.4f ", h_mat[x*n + y]);
		}
		printf("\n");
	} 
	int ur = is_unity_matrix(d_mat,n);
	printf("Is random matrix unit: %d\n",ur);
	ur = is_unity_matrix(d_unity,n);
	printf("Is unit matrix unit: %d\n",ur);
	
	free(h_mat);
	hipFree(d_mat);
	hipFree(d_b);
	hipFree(d_c);
	hipFree(d_unity);
	exit(EXIT_SUCCESS);

}
