#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "random_matrix.h"


/* Kernel that scales up and truncates the random variables */
__global__
void ScaleUp(int n, float *mat, float max, int truncate)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < n && j < n) {
		float x = mat[i*n + j]*max;
		if(truncate)
			x = (float)((int)x);
		mat[i*n + j] = x;
	}
}


float* generate_random_matrix(int n, float max, int truncate)
{
	int size = n*n;

	hiprandGenerator_t gen;
	float *d_mat;

	if(hipMalloc((void **)&d_mat, size*sizeof(float)) != hipSuccess)
	{
		return NULL;
	}

	/* Create pseudo-random number generator */
	if(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) != HIPRAND_STATUS_SUCCESS)
	{
		hipFree(d_mat);
		return NULL;
	}

	/* Set seed */
	if(hiprandSetPseudoRandomGeneratorSeed(gen, 273946962ULL) != HIPRAND_STATUS_SUCCESS)
	{
		hiprandDestroyGenerator(gen);
		hipFree(d_mat);
		return NULL;
	}
	/* Generate n floats on device */
	if(hiprandGenerateUniform(gen, d_mat, size) != HIPRAND_STATUS_SUCCESS)
	{
		hiprandDestroyGenerator(gen);
		hipFree(d_mat);
		return NULL;
	}

	/* Scale and truncate random variables */
	
	/* Let 16 by 16 threads run in parallel per block */
	dim3 threadsPerBlock(16, 16);
	
	int dimx = n / threadsPerBlock.x;
	int dimy = n / threadsPerBlock.y;
	
	/* Is n not divisible by 16 -> increment n by 1 to process the remaining elements */
	if( n > dimx * threadsPerBlock.x)
		dimx++;
	if( n > dimy * threadsPerBlock.y)
		dimy++;
	
	
	dim3 numBlocks(dimx, dimy);
	
	ScaleUp<<<numBlocks, threadsPerBlock>>>(n, d_mat, max, truncate);

	return d_mat;
}
