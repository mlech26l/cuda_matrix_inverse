#include "hip/hip_runtime.h"
/* Matrix Inversion 
 * Group F: M. Lechner, P. Knöbel, J. Lövhall
 *
 * Library to generate a Random Matrix
*/

#include "includes.h"


/* Kernel that scales up and truncates the random variables */
static __global__
void ScaleUp(int n, float *mat, float max, int truncate)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < n && j < n) {
		float x = mat[i*n + j]*max;
		if(truncate)
			x = (float)((int)x);
		mat[i*n + j] = x;
	}
}

/* Allocates an array of size n-by-n on the device
 * and initializies it with random variables.
 * The random variables are in the range of (0, max]
 * If truncate != 0 the digits after the decimal point are truncated
 * i.e. instead of 5.38463 the variable will be 5.0000 
 */
float* random_matrix_generate(int n, float max, int truncate)
{
	int size = n*n;

	hiprandGenerator_t gen;
	float *d_mat;

	if(hipMalloc((void **)&d_mat, size*sizeof(float)) != hipSuccess)
	{
		return NULL;
	}

	/* Create pseudo-random number generator */
	if(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) != HIPRAND_STATUS_SUCCESS)
	{
		hipFree(d_mat);
		return NULL;
	}

	/* Set seed */
	if(hiprandSetPseudoRandomGeneratorSeed(gen, 273946962ULL) != HIPRAND_STATUS_SUCCESS)
	{
		hiprandDestroyGenerator(gen);
		hipFree(d_mat);
		return NULL;
	}
	/* Generate n floats on device */
	if(hiprandGenerateUniform(gen, d_mat, size) != HIPRAND_STATUS_SUCCESS)
	{
		hiprandDestroyGenerator(gen);
		hipFree(d_mat);
		return NULL;
	}

	/* Scale and truncate random variables */
	
	/* Let 16 by 16 threads run in parallel per block */
	dim3 threadsPerBlock(16, 16);
	
	int dimx = n / threadsPerBlock.x;
	int dimy = n / threadsPerBlock.y;
	
	/* Is n not divisible by 16 -> increment n by 1 to process the remaining elements */
	if( n > dimx * threadsPerBlock.x)
		dimx++;
	if( n > dimy * threadsPerBlock.y)
		dimy++;
	
	
	dim3 numBlocks(dimx, dimy);
	
	ScaleUp<<<numBlocks, threadsPerBlock>>>(n, d_mat, max, truncate);

	return d_mat;
}
