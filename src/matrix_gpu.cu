#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "matrix_gpu.h"
#include "testing_util.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


void inverse_gpu(float * in, int size, float * out, int * success){

	float * d_in;
	float * d_out;
	gpuErrchk(hipMalloc((void **)&d_in, size*size* sizeof(float)))
	gpuErrchk(hipMalloc((void **)&d_out, size*size* sizeof(float)))

	gpuErrchk(hipMemcpy(d_in, in, size*size*sizeof(float), hipMemcpyHostToDevice))
	gpuErrchk(hipMemcpy(d_out, out, size*size*sizeof(float), hipMemcpyHostToDevice))

	//Gaussian elimination step
	int i;
	for(i = 0; i < size; i++){
		//todo, there was a check for the possibility to invert here before (row swap), should be brought back adventually

		//scale the row so that [i][i] == 1
		divide_2rows_gpu<<<size/32 + 1,32>>>(i*size + i,d_in , d_in + i*size, d_out + i*size, size);
		//zero out the column below
		zero_out_column_gpu<<<size/32 + 1,32>>>(i, 1, d_in, d_out, size);
	}

	//back substitution step
	int column;
	for(column = size - 1; column >= 1; column--){
		zero_out_column_gpu<<<size/32 + 1,32>>>(column, -1, d_in, d_out, size);
		print_matrix_on_device_kernel<<<1,1>>>(d_in, size);
	}

	//get the inverted matrix back to host memory
	gpuErrchk(hipMemcpy(out, d_out, size*size*sizeof(float), hipMemcpyDeviceToHost))

	*success = 1;
}


__global__
void zero_out_column_gpu(int column, int direction, float * in, float * out, int size){
	int idx = blockIdx.x*blockDim.x  + threadIdx.x;
	if(idx < size){
		int j;
		for(j = column + direction; j < size && j >= 0; j+= direction){

			float scale = in[j*size + column];
			out[idx + j*size] = out[idx+j*size] - (out[idx + column*size] * scale);
			in[idx + j*size] = in[idx+j*size] - (in[idx + column*size] * scale);
		}
	}
}

__global__
void subtract_rows_gpu(int i, float * in, float * out, int size){
	int idx = blockIdx.x*blockDim.x  + threadIdx.x;
	if(idx < size){
		int j;
		for(j = i + 1; j < size; j++){

			float scale = in[j*size + i];
			out[idx + j*size] = out[idx+j*size] - (out[idx + i*size] * scale);
			in[idx + j*size] = in[idx+j*size] - (in[idx + i*size] * scale);
		}
	}
}


//takes vector[denominator_idx] as index and divides all elements in the row from vector and vector2
__global__
void divide_2rows_gpu(int denominator_idx, float * denom_src_vec, float * vector, float * vector2, int size){
	int idx = blockIdx.x*blockDim.x  + threadIdx.x;
	float denominator = denom_src_vec[denominator_idx];

	__syncthreads();
	if(idx < size){
		vector[idx] = vector[idx]/denominator;
		vector2[idx] = vector2[idx]/denominator;
	}
}

__global__
void subtract_row_gpu(float * source, float * target, float scale, int size){
	int idx = blockIdx.x*blockDim.x  + threadIdx.x;
	if(idx < size){
		target[idx] = target[idx] - (source[idx] * scale);
	}
}

__global__
void divide_row_gpu(int denominator_idx, float * denom_src_vec, float * vector, int start_idx, int size){
	int idx = blockIdx.x*blockDim.x  + threadIdx.x;
	float denominator = denom_src_vec[denominator_idx];
	__syncthreads();
	if(idx < size){
		vector[idx + start_idx] = vector[idx + start_idx]/denominator;
	}
}
